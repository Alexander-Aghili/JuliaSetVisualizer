#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <math.h>
#include <stdio.h>
#include "julia_set_cuda.h"

extern "C" {
    #include "perf_man.h"
    #include "complex.h"
    #include "graphics_utilities.h"
}

// Precomputed color values
__device__ uint32_t color_map[16] = {
    0x00421E0F, // get_int_from_color(66, 30, 15)
    0x0019071A, // get_int_from_color(25, 7, 26)
    0x0009012F, // get_int_from_color(9, 1, 47)
    0x00040449, // get_int_from_color(4, 4, 73)
    0x00000764, // get_int_from_color(0, 7, 100)
    0x000C2C8A, // get_int_from_color(12, 44, 138)
    0x001852B1, // get_int_from_color(24, 82, 177)
    0x003987D1, // get_int_from_color(57, 135, 209)
    0x0086B5E5, // get_int_from_color(134, 181, 229)
    0x00D3ECF8, // get_int_from_color(211, 236, 248)
    0x00F1E9BF, // get_int_from_color(241, 233, 191)
    0x00F8C95F, // get_int_from_color(248, 201, 95)
    0x00FFAA00, // get_int_from_color(255, 170, 0)
    0x00CC8000, // get_int_from_color(204, 128, 0)
    0x00995700, // get_int_from_color(153, 87, 0)
    0x006A3403  // get_int_from_color(106, 52, 3)
};

__device__ uint32_t black = 0x00000000;

__device__ uint32_t get_color(int n) {
    if (n < MAX_ITERATIONS && n > 0) {
        int i = n % 16;
        return color_map[i];
    }
    return black;
}

__device__ int color_point(double a, double b, ComplexNumber* c) {
    int n = 0;
    while (n < MAX_ITERATIONS) {
        double u = (a * a - b * b) + c->x;
        double v = (2 * a * b) + c->y;

        if (fabs(u + v) > THRESHOLD) {
            break;
        }

        a = u;
        b = v;

        n++;
    }
    return n;
}

__device__ double screen_map(double input_num, double min_input, double max_input, double min_output, double max_output) {
    return (input_num - min_input) * (max_output - min_output) / (max_input - min_input) + min_output;
}

__global__ void add_pixel_kernel(ComplexBounds* scene_bounds, ComplexNumber* c, uint32_t* image_pixels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < WIDTH && y < HEIGHT) {
        double a = screen_map(x, 0, WIDTH, scene_bounds->min_real, scene_bounds->max_real);
        double b = screen_map(y, 0, HEIGHT, scene_bounds->min_img, scene_bounds->max_img);

        int n = color_point(a, b, c);
        uint32_t color = get_color(n);
        image_pixels[y * WIDTH + x] = color;  // Note: linearized indexing for 2D array
    }
}

void add_pixel(ComplexBounds* scene_bounds, ComplexNumber* c, uint32_t* image_pixels) {
    dim3 threadsPerBlock(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
    dim3 numBlocks((WIDTH + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, (HEIGHT + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);

    add_pixel_kernel<<<numBlocks, threadsPerBlock>>>(scene_bounds, c, image_pixels);
    hipDeviceSynchronize();
}

uint32_t* create_image_pixels_arr(int x, int y) {
    uint32_t *image_pixels;
    hipMallocManaged(&image_pixels, x * y * sizeof(uint32_t));
    return image_pixels;
}

#define NUM_MOVEMENTS 20

void show_julia_start(ComplexScene* scene);
ComplexScene *create_complex_scene(ComplexNumber *c, ComplexBounds *start);

int main(int argc, char** argv) {
    start_sdl();
    ComplexNumber* number = (ComplexNumber*) calloc(1, sizeof(ComplexNumber*));
    ComplexScene *scene = create_complex_scene(number, NULL);

    show_julia_start(scene);
}

ComplexScene *create_complex_scene(ComplexNumber *c, ComplexBounds *start) {
    ComplexScene *scene = (ComplexScene *) calloc(1, sizeof(ComplexScene));
    assert(scene != NULL);
    scene->c = c;


    if (start == NULL) {
        start = (ComplexBounds*) calloc(1, sizeof(ComplexBounds));
        assert(start != NULL);
        start->max_img = DEFAULT_START_MAX_IMG;
        start->min_img = DEFAULT_START_MIN_IMG;
        start->max_real = DEFAULT_START_MAX_REAL;
        start->min_real = DEFAULT_START_MIN_REAL;
    }

    scene->bounds = start;

    return scene;
}


void wait_event(ComplexNumber *c, int* change, int* quit) {
    SDL_Event event;
    struct timeval init;
        gettimeofday(&init,NULL);
        signed long init_time = 1000000 * init.tv_sec + init.tv_usec; 
        while (SDL_PollEvent(&event)) {
            if (event.type == SDL_QUIT) {
                *quit = 1;
                break;
            } else if (event.key.keysym.sym == SDLK_LEFT) {
                c->x -= 0.01;
                *change = 1;
            } else if (event.key.keysym.sym == SDLK_RIGHT) {
                c->x += 0.01;
                *change = 1;
            } else if (event.key.keysym.sym == SDLK_UP) {
                c->y += 0.01;
                *change = 1;
            } else if (event.key.keysym.sym == SDLK_DOWN) {
                c->y -= 0.01;
                *change = 1;
            }
            struct timeval curr;
            gettimeofday(&curr,NULL);
            signed long curr_time = 1000000 * curr.tv_sec + curr.tv_usec; 
            if (curr_time-init_time >= 500) {
               break;
            }
        }
}

void show_julia_start(ComplexScene* scene) {
    int quit = 0;
    int change = 1;

    uint32_t* image_pixels = create_image_pixels_arr(WIDTH, HEIGHT);
    while (!quit) {
        if (change) {
            add_pixel(scene->bounds, scene->c, image_pixels);
            display_image(image_pixels);    
            change = 0;
        }
       wait_event(scene->c, &change, &quit); 
    }
}
