#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <math.h>
#include "julia_set_cuda.h"


__device__ uint32_t color_map[16] = {0x000000, 0x1F1F1F, 0x3F3F3F, 0x5F5F5F, 0x7F7F7F, 0x9F9F9F, 0xBFBFBF, 0xDFDFDF,
                                     0xFF0000, 0xFF7F00, 0xFFFF00, 0x00FF00, 0x0000FF, 0x4B0082, 0x8F00FF, 0xFFFFFF};
__device__ uint32_t black = 0x000000;

__device__ uint32_t get_color(int n) {
    if (n < MAX_ITERATIONS && n > 0) {
        int i = n % 16;
        return color_map[i];
    }
    return black;
}

__device__ int color_point(double a, double b, ComplexNumber* c) {
    int n = 0;
    while (n < MAX_ITERATIONS) {
        double u = (a * a - b * b) + c->x;
        double v = (2 * a * b) + c->y;

        if (fabs(u + v) > THRESHOLD) {
            break;
        }

        a = u;
        b = v;

        n++;
    }
    return n;
}

__device__ double screen_map(double input_num, double min_input, double max_input, double min_output, double max_output) {
    return (input_num - min_input) * (max_output - min_output) / (max_input - min_input) + min_output;
}

__global__ void add_pixel_kernel(ComplexBounds* scene_bounds, ComplexNumber* c, uint32_t* image_pixels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < WIDTH && y < HEIGHT) {
        double a = screen_map(x, 0, WIDTH, scene_bounds->min_real, scene_bounds->max_real);
        double b = screen_map(y, 0, HEIGHT, scene_bounds->min_img, scene_bounds->max_img);

        int n = color_point(a, b, c);
        uint32_t color = get_color(n);
        image_pixels[y * WIDTH + x] = color;  // Note: linearized indexing for 2D array
    }
}

void add_pixel(ComplexBounds* scene_bounds, ComplexNumber* c, uint32_t* image_pixels) {
    dim3 threadsPerBlock(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
    dim3 numBlocks((WIDTH + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, (HEIGHT + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);

    add_pixel_kernel<<<numBlocks, threadsPerBlock>>>(scene_bounds, c, image_pixels);
    hipDeviceSynchronize();
}

uint32_t* create_image_pixels_arr(int x, int y) {
    uint32_t *image_pixels;
    hipMallocManaged(&image_pixels, x * y * sizeof(uint32_t));
    return image_pixels;
}

int main() {
    ComplexBounds scene_bounds = { -2.0, 1.0, -1.5, 1.5 };
    ComplexNumber c = { -0.7, 0.27015 };

    uint32_t* image_pixels = create_image_pixels_arr(WIDTH, HEIGHT);

    add_pixel(&scene_bounds, &c, image_pixels);

    hipFree(image_pixels);
    return 0;
}
