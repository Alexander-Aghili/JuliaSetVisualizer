#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <math.h>
#include <stdio.h> 
#include <unistd.h>
#include "julia_set_cuda.h"

extern "C" {
    #include "perf_man.h"
    #include "complex.h"
    #include "graphics_utilities.h"
}

// Precomputed color values
__device__ uint32_t color_map[16] = {
    0x00421E0F, // get_int_from_color(66, 30, 15)
    0x0019071A, // get_int_from_color(25, 7, 26)
    0x0009012F, // get_int_from_color(9, 1, 47)
    0x00040449, // get_int_from_color(4, 4, 73)
    0x00000764, // get_int_from_color(0, 7, 100)
    0x000C2C8A, // get_int_from_color(12, 44, 138)
    0x001852B1, // get_int_from_color(24, 82, 177)
    0x003987D1, // get_int_from_color(57, 135, 209)
    0x0086B5E5, // get_int_from_color(134, 181, 229)
    0x00D3ECF8, // get_int_from_color(211, 236, 248)
    0x00F1E9BF, // get_int_from_color(241, 233, 191)
    0x00F8C95F, // get_int_from_color(248, 201, 95)
    0x00FFAA00, // get_int_from_color(255, 170, 0)
    0x00CC8000, // get_int_from_color(204, 128, 0)
    0x00995700, // get_int_from_color(153, 87, 0)
    0x006A3403  // get_int_from_color(106, 52, 3)
};

__device__ uint32_t black = 0x00000000;

__device__ uint32_t get_color(int n) {
    if (n < MAX_ITERATIONS && n > 0) {
        int i = n % 16;
        return color_map[i];
    }
    return black;
}

__device__ int color_point(double a, double b, ComplexNumber* c) {
    int n = 0;
    while (n < MAX_ITERATIONS) {
        double u = (a * a - b * b) + c->x;
        double v = (2 * a * b) + c->y;

        if (fabs(u + v) > THRESHOLD) {
            break;
        }

        a = u;
        b = v;

        n++;
    }
    return n;
}

__device__ double screen_map(double input_num, double min_input, double max_input, double min_output, double max_output) {
    return (input_num - min_input) * (max_output - min_output) / (max_input - min_input) + min_output;
}

__global__ void add_pixel_kernel(ComplexBounds* scene_bounds, ComplexNumber* c, uint32_t* image_pixels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < WIDTH && y < HEIGHT) {
        double a = screen_map(x, 0, WIDTH, scene_bounds->min_real, scene_bounds->max_real);
        double b = screen_map(y, 0, HEIGHT, scene_bounds->min_img, scene_bounds->max_img);

        int n = color_point(a, b, c);
        uint32_t color = get_color(n);
        image_pixels[y * WIDTH + x] = color;  // Note: linearized indexing for 2D array
    }
}

void add_pixel(ComplexBounds* scene_bounds, ComplexNumber* c, uint32_t* image_pixels) {
    dim3 threadsPerBlock(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
    dim3 numBlocks((WIDTH + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, (HEIGHT + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);

    add_pixel_kernel<<<numBlocks, threadsPerBlock>>>(scene_bounds, c, image_pixels);
    hipDeviceSynchronize();
}

uint32_t* create_image_pixels_arr(int x, int y) {
    uint32_t *image_pixels;
    hipMallocManaged(&image_pixels, x * y * sizeof(uint32_t));
    return image_pixels;
}

#define NUM_MOVEMENTS 20

void show_julia_start(ComplexScene* scene);
ComplexScene *create_complex_scene(ComplexNumber *c, ComplexBounds *start);

int main(int argc, char** argv) {
    start_sdl();
    ComplexNumber* number = (ComplexNumber*) calloc(1, sizeof(ComplexNumber*));
    ComplexScene *scene = create_complex_scene(number, NULL);

    show_julia_start(scene);
}

ComplexBounds* get_start_bounds() {
    ComplexBounds* start = (ComplexBounds*) calloc(1, sizeof(ComplexBounds));
    assert(start != NULL);
    start->max_img = DEFAULT_START_MAX_IMG;
    start->min_img = DEFAULT_START_MIN_IMG;
    start->max_real = DEFAULT_START_MAX_REAL;
    start->min_real = DEFAULT_START_MIN_REAL;
    return start;

}

ComplexScene *create_complex_scene(ComplexNumber *c, ComplexBounds *start) {
    ComplexScene *scene = (ComplexScene *) calloc(1, sizeof(ComplexScene));
    assert(scene != NULL);
    scene->c = c;


    if (start == NULL) {
        start = get_start_bounds();
    }

    scene->bounds = start;

    return scene;
}

void zoom(ComplexBounds* bounds, double scaling_factor) {
    double real_center = (bounds->max_real + bounds->min_real)/2;
    double img_center = (bounds->max_img + bounds->min_img)/2;
    bounds->max_real = real_center + scaling_factor * (bounds->max_real - real_center);
    bounds->min_real = real_center + scaling_factor * (bounds->min_real - real_center);
    bounds->max_img = img_center + scaling_factor * (bounds->max_img - img_center);
    bounds->min_img = img_center + scaling_factor * (bounds->min_img - img_center);
}

int wait_event(ComplexScene *scene, int* change, int* quit) {
    SDL_Event event;
    ComplexNumber* c = scene->c;
    ComplexBounds* bounds = scene->bounds;
    int ret = 0;
    struct timeval init;
    gettimeofday(&init,NULL);
    signed long init_time = 1000000 * init.tv_sec + init.tv_usec; 
    double move_amount = 0.005;
    while (SDL_PollEvent(&event)) {
        if (event.type == SDL_QUIT) {
            *quit = 1;
            break;
        } else if (event.key.keysym.sym == SDLK_LEFT) {
            c->x -= move_amount;
            *change = 1;
        } else if (event.key.keysym.sym == SDLK_RIGHT) {
            c->x += move_amount;
            *change = 1;
        } else if (event.key.keysym.sym == SDLK_UP) {
            c->y += move_amount;
            *change = 1;
        } else if (event.key.keysym.sym == SDLK_DOWN) {
            c->y -= move_amount;
            *change = 1;
        } else if (event.button.type == SDL_MOUSEBUTTONDOWN && event.button.button == SDL_BUTTON_LEFT) {
            int x1, y1, x2, y2;
            SDL_GetMouseState(&x1, &y1);

            SDL_Event mouse_up;
            mouse_up.button.type = SDL_MOUSEBUTTONUP;
            while (SDL_WaitEvent(&mouse_up)) {
                if (mouse_up.button.type == SDL_MOUSEBUTTONUP) {
                    break;
                }
            }

            SDL_GetMouseState(&x2, &y2);

            double xdiff = ((double)(x1 - x2)/(sqrt((double)(x1*x1)+(double)(x2*x2))))*((bounds->max_real-bounds->min_real)/**(bounds->max_img-bounds->min_img)*/);
            double ydiff = ((double)(y1 - y2)/(sqrt((double)(y1*y1)+(double)(y2*y2))))*((bounds->max_real-bounds->min_real)/**(bounds->max_img-bounds->min_img)*/);

            bounds->max_real += xdiff;
            bounds->min_real += xdiff;
            bounds->max_img += ydiff;
            bounds->min_img += ydiff;
            *change = 1;
        } else if (event.key.keysym.sym == SDLK_EQUALS) {
            zoom(bounds, 0.9);
            *change = 1;
        } else if (event.key.keysym.sym == SDLK_MINUS) {
            zoom(bounds, 1.1);
            *change = 1;
        } else if (event.key.keysym.sym == SDLK_RETURN) {
            printf("(%f, %f, %f, %f)\n", bounds->max_real, bounds->min_real, bounds->max_img, bounds->min_img);
            ret = 1;
        }
        /*            
                      struct timeval curr;
                      gettimeofday(&curr,NULL);
                      signed long curr_time = 1000000 * curr.tv_sec + curr.tv_usec; 
                      if (curr_time-init_time >= 500) {
                      break;
                      }
         */
    }
    return ret;
}

void zoom_display(ComplexScene* scene);

void show_julia_start(ComplexScene* scene) {
    int quit = 0;
    int change = 1;

    uint32_t* image_pixels = create_image_pixels_arr(WIDTH, HEIGHT);
    while (!quit) {
        if (change) {
            add_pixel(scene->bounds, scene->c, image_pixels);
            display_image(image_pixels);    
            change = 0;
        }
       int ret = wait_event(scene, &change, &quit); 
       if (ret == 1) {
           //Zoom 
           zoom_display(scene);
       }
    }
}

void zoom_display(ComplexScene* scene) {
    ComplexBounds* start = get_start_bounds(); 
    ComplexBounds* end = scene->bounds;

    int frames = 10;
    double max_img_incr = (end->max_img - start->max_img) / frames;
    double min_img_incr = (end->min_img - start->min_img) / frames;
    double max_real_incr = (end->max_real - start->max_real) / frames;
    double min_real_incr = (end->min_real - start->min_real) / frames;

    double curr_max_img = start->max_img;
    double curr_min_img = start->min_img;
    double curr_max_real = start->max_real;
    double curr_min_real = start->min_real;

    uint32_t** image_list = (uint32_t**) calloc(1000, sizeof(uint32_t*));
    
    int i = 0;
    while (fabs(curr_max_img - end->max_img) > EPSILON
           || fabs(curr_max_real - end->max_real) > EPSILON
           || fabs(curr_min_img - end->min_img) > EPSILON
           || fabs(curr_min_real - end->min_real) > EPSILON) {
        
        uint32_t* image_pixels = create_image_pixels_arr(WIDTH, HEIGHT);
        image_list[i++] = image_pixels;

        add_pixel(start, scene->c, image_pixels);

        curr_max_img += max_img_incr;
        curr_min_img += min_img_incr;
        curr_max_real += max_real_incr;
        curr_min_real += min_real_incr;

        start->max_img = curr_max_img;
        start->min_img = curr_min_img;
        start->max_real = curr_max_real;
        start->min_real = curr_min_real;
           
        max_img_incr *= SCALE_FACTOR;
        min_img_incr *= SCALE_FACTOR;
        max_real_incr *= SCALE_FACTOR;
        min_real_incr *= SCALE_FACTOR;
    }

    for (int k = 0; k < i; k++) {
        display_image(image_list[k]);
        usleep(100);
    }

}
